#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <iostream>

#define checkCUDA(expression)                                                  \
  {                                                                            \
    hipError_t status = (expression);                                         \
    if (status != hipSuccess) {                                               \
      std::cerr << "Error on line " << __LINE__ << ": "                        \
                << hipGetErrorString(status) << std::endl;                    \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  }

struct DepthwiseArgs {
  // Input layer dimensions
  int batch;
  int in_rows;
  int in_cols;
  int in_depth;
  int filter_rows;
  int filter_cols;
  int depth_multiplier;
  int stride;
  int pad_rows; // Amount of padding to the top of the input
  int pad_cols; // Amount of padding to the left of the input

  // Output layer dimensions
  int out_rows;
  int out_cols;
  int out_depth;

  DepthwiseArgs()
      : batch(0), in_rows(0), in_cols(0), in_depth(0), filter_rows(0),
        filter_cols(0), depth_multiplier(0), stride(0), pad_rows(0),
        pad_cols(0), out_rows(0), out_cols(0), out_depth(0) {}
};

template <typename T> class GpuGridRange {
  struct Iterator {
    __device__ Iterator(T index, T delta) : index_(index), delta_(delta) {}
    __device__ T operator*() const { return index_; }
    __device__ Iterator &operator++() {
      index_ += delta_;
      return *this;
    }
    __device__ bool operator!=(const Iterator &other) const {
      bool greater = index_ > other.index_;
      bool less = index_ < other.index_;
      // Anything past an end iterator (delta_ == 0) is equal.
      // In range-based for loops, this optimizes to 'return less'.
      if (!other.delta_) {
        return less;
      }
      if (!delta_) {
        return greater;
      }
      return less || greater;
    }

  private:
    T index_;
    const T delta_;
  };

public:
  __device__ GpuGridRange(T begin, T delta, T end)
      : begin_(begin), delta_(delta), end_(end) {}

  __device__ Iterator begin() const { return Iterator{begin_, delta_}; }
  __device__ Iterator end() const { return Iterator{end_, 0}; }

private:
  T begin_;
  T delta_;
  T end_;
};

template <typename T> __device__ GpuGridRange<T> GpuGridRangeX(T count) {
  return GpuGridRange<T>(
      /*begin=*/blockIdx.x * blockDim.x + threadIdx.x,
      /*delta=*/gridDim.x * blockDim.x, /*end=*/count);
}

template <typename T> __host__ __device__ T GpuLdg(const T *address) {
  return __ldg(address);
}

template <typename T> __host__ __device__ inline T ldg(const T *ptr) {
  return GpuLdg(ptr);
}

#define GPU_1D_KERNEL_LOOP(i, n) for (int i : GpuGridRangeX<int>(n))
#define UNROLL _Pragma("unroll")
#define NOUNROLL _Pragma("nounroll")

template <typename T> struct CudaSupportedTypeImpl { using type = T; };

template <typename T>
using CudaSupportedType = typename CudaSupportedTypeImpl<T>::type;

template <typename T>
__device__ CudaSupportedType<T> *ToCudaSupportedPtr(T *ptr) {
  return reinterpret_cast<CudaSupportedType<T> *>(ptr);
}

template <typename From, typename To>
using ToTypeIfConvertible =
    typename std::enable_if<std::is_convertible<From, To>::value, To>::type;

template <typename T, typename U>
__device__ ToTypeIfConvertible<U, T> GpuAtomicAdd(T *ptr, U value) {
  return atomicAdd(ToCudaSupportedPtr(ptr), value);
  // return *ptr = value;
}

template <typename T>
__global__ void __launch_bounds__(512, 2)
    DepthwiseConv2dBackwardFilterKernel(const DepthwiseArgs args,
                                        const T *__restrict__ out_backprop,
                                        const T *__restrict__ input,
                                        T *__restrict__ filter_backprop) {
  const int batch_num = args.batch;
  const int in_channel = args.in_depth;
  const int in_height = args.in_rows;
  const int in_width = args.in_cols;
  const int filter_width = args.filter_cols;
  const int stride_height = args.stride;
  const int stride_width = args.stride;
  const int pad_height = args.pad_rows;
  const int pad_width = args.pad_cols;
  const int out_channel = args.out_depth;
  const int out_height = args.out_rows;
  const int out_width = args.out_cols;

  typedef hipcub::WarpReduce<T> WarpReduce;
  typename WarpReduce::TempStorage temp_storage;

  T partial_sum = 0.;

  const int filter_w = blockIdx.x;
  const int filter_h = blockIdx.y;
  const int out_c = blockIdx.z;
  assert(gridDim.x == filter_width);
  assert(gridDim.z == out_channel);
  const int filter_backprop_offset =
      ((filter_h * filter_width) + filter_w) * out_channel + out_c;
  const int out_spatial_size = out_height * out_width;

  for (int batch = 0; batch < batch_num; batch++) {
    const int input_offset_temp = (batch * in_channel + out_c) * in_height;
    const int output_backprop_offset_temp =
        (batch * out_channel + out_c) * out_height;
    for (int i = threadIdx.x; i < out_spatial_size; i += blockDim.x) {
      const int out_col = i % out_width;
      const int out_row = i / out_width;
      // We use the formula: `(in_row - filter_w + pad_left ) / stride =
      // out_row` to compute corresponding in_row and out_row positions. Similar
      // for in_col and out_col.
      const int in_row = out_row * stride_height + filter_h - pad_height;
      const int in_col = out_col * stride_width + filter_w - pad_width;

      if (in_row < 0 || in_col < 0 || in_row >= in_height ||
          in_col >= in_width) {
        continue;
      }

      int input_offset = (input_offset_temp + in_row) * in_width + in_col;
      int output_backprop_offset =
          (output_backprop_offset_temp + out_row) * out_width + out_col;
      partial_sum += out_backprop[output_backprop_offset] * input[input_offset];
    }
  }

  T val = WarpReduce(temp_storage).Sum(partial_sum);
  if (hipcub::LaneId() == 0) {
    T *addr = filter_backprop + filter_backprop_offset;
    GpuAtomicAdd(addr, val);
  }
}

template <typename T> void init_array(T *dev_ptr, int n) {
  T *host_ptr = new T[n];
  for (int i = 0; i < n; i++) {
    host_ptr[i] = 0.1;
  }
  checkCUDA(
      hipMemcpy(dev_ptr, host_ptr, sizeof(T) * n, hipMemcpyHostToDevice));
  delete[] host_ptr;
}

template <typename T>
void print_array(T *dev_ptr, int n, const std::string &prompt) {
  std::cout << prompt << std::endl;
  T *host_ptr = new T[n];
  checkCUDA(
      hipMemcpy(host_ptr, dev_ptr, sizeof(T) * n, hipMemcpyDeviceToHost));
  for (int i = 0; i < n; i++) {
    printf("%f, ", static_cast<float>(host_ptr[i]));
    if ((i + 1) % 10 == 0)
      break;
  }
  printf("\n");
  delete[] host_ptr;
}

inline int DivUp(int a, int b) { return (a + b - 1) / b; }

int main(int argc, char **argv) {

  int dargs[13] = {3, 128, 128, 144, 3, 3, 1, 1, 1, 1, 128, 128, 144};

  if (argc > 6) {
    dargs[0] = atoi(argv[1]);
    dargs[1] = atoi(argv[2]);
    dargs[2] = atoi(argv[3]);
    dargs[3] = atoi(argv[4]);
    dargs[4] = atoi(argv[5]);
    dargs[5] = atoi(argv[6]);
    dargs[10] = atoi(argv[2]);
    dargs[11] = atoi(argv[3]);
    dargs[12] = atoi(argv[4]);
  }
  printf("XXX N,H,W,C,R,S: %d %d %d %d %d %d\n", dargs[0], dargs[1], dargs[2],
         dargs[3], dargs[4], dargs[5]);

  DepthwiseArgs args;
  args.batch = dargs[0];
  args.in_rows = dargs[1];
  args.in_cols = dargs[2];
  args.in_depth = dargs[3];
  args.filter_rows = dargs[4];
  args.filter_cols = dargs[5];
  args.depth_multiplier = dargs[6];
  args.stride = dargs[7];
  args.pad_rows = dargs[8];
  args.pad_cols = dargs[9];
  args.out_rows = dargs[10];
  args.out_cols = dargs[11];
  args.out_depth = dargs[12];

  int num_out_backprop =
      args.batch * args.out_depth * args.out_rows * args.out_cols;
  int num_input = args.batch * args.in_depth * args.in_rows * args.in_cols;
  int num_filter_backprop = args.in_depth * args.filter_rows * args.filter_cols;

  int out_backprop_bytes = sizeof(float) * num_out_backprop;
  int input_bytes = sizeof(float) * num_input;
  int filter_backprop_bytes = sizeof(float) * num_filter_backprop;

  float *out_backprop;
  float *input;
  float *filter_backprop;

  checkCUDA(hipMalloc(&out_backprop, out_backprop_bytes));
  checkCUDA(hipMalloc(&input, input_bytes));
  checkCUDA(hipMalloc(&filter_backprop, filter_backprop_bytes));

  init_array(out_backprop, num_out_backprop);
  init_array(input, num_input);

  auto device_fn = DepthwiseConv2dBackwardFilterKernel<float>;
  dim3 blocks = dim3(args.filter_cols, args.filter_rows, args.out_depth);
  dim3 threads = dim3(512, 1, 1);
  printf("XXX blocks: %d %d %d\n", blocks.x, blocks.y, blocks.z);
  printf("XXX threads: %d %d %d\n", threads.x, threads.y, threads.z);

  auto launcher = [&](int repeats) {
    for (int i = 0; i < repeats; i++) {
      device_fn<<<blocks, threads>>>(args, out_backprop, input,
                                     filter_backprop);
    }
  };
  // warmup
  launcher(20);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  const int repeats = 50;
  launcher(repeats);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("XXX time(ms): %f\n", milliseconds / repeats);

  print_array(filter_backprop, num_filter_backprop, "Results:");
}
