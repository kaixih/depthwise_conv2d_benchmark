#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "int_divider.h"

#define checkCUDA(expression)                              \
{                                                          \
  hipError_t status = (expression);                       \
  if (status != hipSuccess) {                             \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipGetErrorString(status) << std::endl;  \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}
__global__ void test_division_kernel(const int* numerators, int size_0,
                                     const int* divisors, int size_1,
                                     bool* result) {
  bool pass = true;
  for (int i = 0; i < size_0; i++) {
    for (int j = 0; j < size_1; j++) {
      int ref_div = numerators[i] / divisors[j];
      int ref_mod = numerators[i] % divisors[j];

      FastDividerUint32 divisor(divisors[j]);
      int new_div = numerators[i] / divisor;
      int new_mod = numerators[i] % divisor;
      if (ref_div != new_div || ref_mod != new_mod) {
        printf("Reference: %d /(%%) %d = %d(%d) But, we got: %d(%d)\n",
               numerators[i], divisors[i], ref_div, ref_mod, new_div, new_mod);
        pass = false;
        break;
      }
    }
  }
  *result = pass;
}

bool test_division(const int* numerators, int size_0, const int* divisors,
                   int size_1) {
  bool pass = true;
  for (int i = 0; i < size_0; i++) {
    for (int j = 0; j < size_1; j++) {
      int ref_div = numerators[i] / divisors[j];
      int ref_mod = numerators[i] % divisors[j];

      FastDividerUint32 divisor(divisors[j]);
      int new_div = numerators[i] / divisor;
      int new_mod = numerators[i] % divisor;
      if (ref_div != new_div || ref_mod != new_mod) {
        printf("Reference: %d /(%%) %d = %d(%d) But, we got: %d(%d)\n",
               numerators[i], divisors[i], ref_div, ref_mod, new_div, new_mod);
        pass = false;
        break;
      }
    }
  }
  return pass;
}

int main() {
  const int count = 100;
  int* general_numerators = new int[count];
  int special_numerators[] = {0, 1, INT32_MAX};
  int* general_divisors = new int[count];
  int special_divisors[] = {1, INT32_MAX};
  for (int i = 0; i < count; i++) {
    general_numerators[i] = rand() % INT32_MAX + 1;
    general_divisors[i] = rand() % INT32_MAX + 1;
  }
  // CPU
  bool pass = test_division(general_numerators, count, general_divisors, count);
  printf("CPU Test general_numerators/general_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  pass = test_division(general_numerators, count, special_divisors, 2);
  printf("CPU Test general_numerators/special_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  pass = test_division(special_numerators, 3, general_divisors, count);
  printf("CPU Test special_numerators/general_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  pass = test_division(special_numerators, 3, special_divisors, 2);
  printf("CPU Test special_numerators/special_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));

  // GPU
  int* d_general_numerators;
  int* d_special_numerators;
  int* d_general_divisors;
  int* d_special_divisors;
  bool* d_result;
  checkCUDA(hipMalloc((void**)(&d_general_numerators), sizeof(int) * count));
  checkCUDA(hipMalloc((void**)(&d_special_numerators), sizeof(int) * 3));
  checkCUDA(hipMalloc((void**)(&d_general_divisors), sizeof(int) * count));
  checkCUDA(hipMalloc((void**)(&d_special_divisors), sizeof(int) * 2));
  checkCUDA(hipMalloc((void**)(&d_result), sizeof(bool)));
  checkCUDA(hipMemcpy(d_general_numerators, general_numerators,
                       sizeof(int) * count, hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(d_special_numerators, special_numerators,
                       sizeof(int) * 3, hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(d_general_divisors, general_divisors,
                       sizeof(int) * count, hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(d_special_divisors, special_divisors,
                       sizeof(int) * 2, hipMemcpyHostToDevice));
  checkCUDA(hipMemset(d_result, 0, sizeof(bool)));

  test_division_kernel<<<1, 1>>>(d_general_numerators, count,
                                 d_general_divisors, count, d_result);
  checkCUDA(hipMemcpy(&pass, d_result, sizeof(bool), hipMemcpyDeviceToHost));
  printf("GPU Test general_numerators/general_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  test_division_kernel<<<1, 1>>>(d_general_numerators, count,
                                 d_special_divisors, 2, d_result);
  checkCUDA(hipMemcpy(&pass, d_result, sizeof(bool), hipMemcpyDeviceToHost));
  printf("GPU Test general_numerators/special_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  test_division_kernel<<<1, 1>>>(d_special_numerators, 3,
                                 d_general_divisors, count, d_result);
  checkCUDA(hipMemcpy(&pass, d_result, sizeof(bool), hipMemcpyDeviceToHost));
  printf("GPU Test special_numerators/general_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
  test_division_kernel<<<1, 1>>>(d_special_numerators, 3,
                                 d_special_divisors, 2, d_result);
  checkCUDA(hipMemcpy(&pass, d_result, sizeof(bool), hipMemcpyDeviceToHost));
  printf("GPU Test special_numerators/special_divsors %s!\n",
         (pass ? "PASSED" : "FAILED"));
}

